#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/layers/conv_layer.hpp"
#include "caffe/adaptive_probabilistic_pruning.hpp"
#define SHOW_INTERVAL 10

using namespace std;
namespace caffe {

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
          
    /// ADDED BY WANGHUAN -----------------------------------
    Dtype* muweight = this->blobs_[0]->mutable_cpu_data();
    const int count = this->blobs_[0]->count();
    const int num_row = this->blobs_[0]->shape()[0];
    const int num_col = count / num_row;
    const string layer_name = this->layer_param_.name();
    const int L = APP::layer_index[layer_name];
    const string mthd = APP::prune_method;
    this->IF_restore = false;
    
    // IF_mask
    const bool IF_prune       = APP::prune_method != "None";
    const bool IF_enough_iter = (APP::step_ - 1) >= APP::prune_iter_begin;
    const bool IF_pruned      = this->pruned_ratio > 0;
    const bool IF_mask        = IF_prune && (IF_enough_iter || IF_pruned);
    
    if (this->phase_ == TRAIN) {
        if (IF_mask) {
            /*
            if (mthd == "CP" && L != APP::layer_cnt) {
                if (APP::step_ - 2 <= APP::iter_prune_finished[L + 1]) {
                    UpdateNumPrunedRow();
                }
            } else if ((mthd == "FP" || mthd == "TP") && L != 0) {
                if (APP::step_ - 1 <= APP::iter_prune_finished[L - 1]) {
                    UpdateNumPrunedCol();
                }
            } else if (mthd == "PPc" && L != APP::layer_cnt && APP::iter_prune_finished[L] != INT_MAX) {
                if (APP::step_ - 2 <= APP::iter_prune_finished[L + 1]) {
                    UpdateNumPrunedRow();
                } else if (APP::IF_never_updated[L]) {
                    UpdateNumPrunedRow();
                }
            } else if (mthd == "PPr" && L != 0 && APP::iter_prune_finished[L] != INT_MAX) {
                if (APP::step_ - 1 <= APP::iter_prune_finished[L - 1]) {
                    UpdateNumPrunedCol();
                } else if (APP::IF_never_updated[L]) {
                    UpdateNumPrunedCol();
                }
            }
            */
            
            this->pruned_ratio = 1 - (1 - APP::num_pruned_col[L] * 1.0 / num_col) * (1 - APP::num_pruned_row[L] * 1.0 / num_row);
            if (APP::iter_prune_finished[L] == -1) {
                if (this->pruned_ratio >= this->prune_ratio) {
                    if (mthd.substr(0, 2) == "PP") { CleanWorkForPP(); } // last time, do some clean work
                    APP::iter_prune_finished[L] = APP::step_ - 1;
                    cout << layer_name << " prune finished!" 
                         << "  step: " << APP::step_
                         << "  pruned_ratio: " << this->pruned_ratio << endl;
                }
            }
        }
        
        // Print and check
        if (mthd != "None" && L < 5 && APP::inner_iter == 0) {
            cout << layer_name << "  IF_mask: " << IF_mask 
                 << "  pruned_ratio: " << this->pruned_ratio
                 << "  prune_ratio: " << this->prune_ratio 
                 << "  num_pruned_col: " << APP::num_pruned_col[L]
                 << "  num_pruned_row: " << APP::num_pruned_row[L] << endl;
        }
        
        // Print and check (before pruning)
        if (L == 1 && APP::step_ % SHOW_INTERVAL == 0 && APP::inner_iter == 0) {
            /// cout.setf(std::ios::left);
            cout.width(5);  cout << "Index" << "   ";
            cout.width(18); cout << "WeightBeforeMasked" << "   ";
            cout.width(4);  cout << "Mask" << "   ";
            cout.width(4);  cout << "Prob" << endl;
            for (int i = 0; i < 20; ++i) {
                cout.width(3);  cout << "#";
                cout.width(2);  cout << i+1 << "   ";
                if (mthd == "PPr" || mthd == "FP" || mthd == "TP") { /// i denotes row
                    cout.width(18); cout << muweight[i * num_col] << "   ";
                    cout.width(4);  cout << this->masks_[i * num_col] << "   ";
                    cout.width(4);  cout << APP::history_prob[L][i] << endl;
                } else { /// i denotes column
                    cout.width(18); cout << muweight[i] << "   ";
                    cout.width(4);  cout << this->masks_[i] << "   ";
                    cout.width(4);  cout << APP::history_prob[L][i] << endl;
                }
            }
        }

        // Update masks and apply masks
        if (IF_mask && this->pruned_ratio < this->prune_ratio) {
            if (mthd == "CP" && APP::criteria == "L2-norm") {
                /// ColumnPrune();
            } else if (mthd == "FP") {
                if ((APP::step_ - 1) % GetPruneInterval() == 0) { FilterPrune(); }    
            } else if (mthd == "PPc" && IF_hppf()) {
                ProbPruneCol();
            } else if (mthd == "PPr" && IF_hppf()) {
                ProbPruneRow();
            }else if (mthd == "TP") {
                for (int i = 0; i < count; ++i) {
                    muweight[i] *= this->masks_[i]; 
                }  /// explictly prune, because seems TP is wrong somewhere.
            }
        }  
   
        
        // Logging
        if (APP::num_log) {
            const int num_log = APP::log_index[L].size();
            for (int k = 0; k < num_log; ++k) {
                const int index = APP::log_index[L][k]; 
                Dtype sum = 0;
                for (int i = 0; i < num_row; ++i) {
                    sum += fabs(muweight[i * num_col + index]);
                }
                sum /= num_row;
                APP::log_weight[L][k].push_back(sum);
            }
        }
        
    } else {
        if (mthd == "PPc") {
            Dtype rands[num_col];
            caffe_rng_uniform(num_col, (Dtype)0, (Dtype)1, rands);
            for (int i = 0; i < count; ++i) {
                this->masks_[i] = rands[i % num_col] < APP::history_prob[L][i % num_col] ? 1 : 0; /// generate masks
            }
            for (int i = 0; i < count; ++i) { this->weight_backup[i] = muweight[i]; } /// backup weights
            this->IF_restore = true;
            for (int i = 0; i < count; ++i) { muweight[i] *= this->masks_[i]; } /// do pruning
            
        } else if (mthd == "PPr") {
            Dtype rands[num_row];
            caffe_rng_uniform(num_row, (Dtype)0, (Dtype)1, rands);
            for (int i = 0; i < count; ++i) {
                this->masks_[i] = rands[i / num_col] < APP::history_prob[L][i / num_col] ? 1 : 0; /// generate masks
            }              
            for (int i = 0; i < count; ++i) { this->weight_backup[i] = muweight[i]; } /// backup weights
            this->IF_restore = true;
            for (int i = 0; i < count; ++i) { muweight[i] *= this->masks_[i]; } /// do pruning
            
        }
    }
    
    
  /// ------------------------------------------------------
    const Dtype* weight = this->blobs_[0]->gpu_data();
    for (int i = 0; i < bottom.size(); ++i) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Dtype* top_data = top[i]->mutable_gpu_data();
        for (int n = 0; n < this->num_; ++n) {
            this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
            top_data + n * this->top_dim_);
            if (this->bias_term_) {
                const Dtype* bias = this->blobs_[1]->gpu_data();
                this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
            }
        }
    }
    /// this->bottom_dim_: bottom feature map size, input
    /// this->top_dim_: top feature map size, output
    /// this->num_: batch size
    
    /// Print feature map to check --------
    /// If row 3 and 8 are pruned in previous layer, then channel 3 and 8 will be only biases in this layer's feature map.
    /**
    if (!APP::IN_TEST && L == 0) {
        cout << "bottom.size(): " << bottom.size() << endl;
        for (int i = 0; i < bottom.size(); ++i) {
            const Dtype* top_data = top[i]->cpu_data();
            const int channel = top[i]->shape()[1];
            const int width   = top[i]->shape()[2];
            const int height  = top[i]->shape()[3];
            cout << "channel: " << channel << " " << width << " " <<  height << endl;
            
            vector<Dtype> sum(channel, 0);
            for (int c = 0; c < channel; ++c) {
                for (int w = 0 ; w < width; ++w) {
                    for (int h = 0; h < height; ++h) {
                        sum[c] += fabs(top_data[0 + c * width * height + w * height + h]);
                    }
                }
            }
            for (int c = 0; c < channel; ++c) {
                cout << sum[c] << "  ";
            }
            cout << endl;
        }
    }
    */
    /// -----------------------------------
    
    
    
    /// Restore weights ----------------
    if (this->IF_restore) {
        /// cout << layer_name << ": restore weights! " << endl;
        this->blobs_[0]->mutable_cpu_data();
        /// this->blobs_[0]->gpu_data(); 
        /// Interesting! If the above line is added, something like "control" seems to transfer from cpu to gpu. 
        /// Then modifying cpu weights won't affect their gpu counterparts.
        for (int i = 0; i < count; ++i) {
            muweight[i] = this->weight_backup[i];
        }
        
        /**
        /// ========================
        /// Chech restore
        cout << "weights from cpu:" << endl;
        for (int i = 0; i < 20; ++i) {
            cout << muweight[i] << " ";
        }
        cout << endl;

        Dtype weight_cpu[count];
        const Dtype* weight_gpu = this->blobs_[0]->gpu_data();
        cout << "weights copied from gpu:" << endl;
        hipMemcpy(weight_cpu, weight_gpu, sizeof(Dtype) * count, hipMemcpyDeviceToHost);
        for (int i = 0; i < 20; ++i) {
            cout << weight_cpu[i] << " ";
        }
        cout << endl;
        /// ========================
        */
    }
    /// --------------------------------
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
  
/// ADDED BY WANGHUAN ------------------------------------------
    Dtype* muweight_diff = this->blobs_[0]->mutable_cpu_diff();      
    const int count = this->blobs_[0]->count();
    const int num_row = this->blobs_[0]->shape()[0];
    const int num_col = count / num_row;
    const int L = APP::layer_index[this->layer_param_.name()];
    const string mthd = APP::prune_method;
    
    /// UpdateDiffs(); /// update second diff and so on

    /// Print and check
    if (L == 1 && APP::step_ % SHOW_INTERVAL == 0 && APP::inner_iter == 0) {
        cout.width(5);  cout << "Index" << "   ";
        cout.width(16); cout << "DiffBeforeMasked" << "   ";
        cout.width(4);  cout << "Mask" << "   ";
        cout.width(4);  cout << "Prob" << endl;
        for (int i = 0; i < 20; ++i) {
            cout.width(3);  cout << "#";
            cout.width(2);  cout << i+1 << "   ";
            if (mthd == "PPr" || mthd == "FP" || mthd == "TP") { /// i denotes row
                cout.width(16); cout << muweight_diff[i * num_col] << "   ";
                cout.width(4);  cout << this->masks_[i * num_col] << "   ";
                cout.width(4);  cout << APP::history_prob[L][i] << endl;
            } else { /// i denotes column
                cout.width(16); cout << muweight_diff[i] << "   ";
                cout.width(4);  cout << this->masks_[i] << "   ";
                cout.width(4);  cout << APP::history_prob[L][i] << endl;
            }
        }
    }
    
    /// Diff log
    if (APP::num_log) {
        const int num_log = APP::log_index[L].size();
        for (int i = 0; i < num_log; ++i) {
            const int index = APP::log_index[L][i];
            Dtype sum = 0;
            for (int r = 0; r < num_row; ++r) {
                sum += fabs(muweight_diff[r * num_col + index]);
            }
            sum /= num_row;
            APP::log_diff[L][i].push_back(sum);
        }
    }
    

    /// IF_mask
    const bool IF_prune       = mthd != "None";
    const bool IF_enough_iter = (APP::step_ - 1) >= APP::prune_iter_begin;
    const bool IF_pruned      = this->pruned_ratio > 0;
    const bool IF_mask        = IF_prune && (IF_enough_iter || IF_pruned) ;
    if (IF_mask) {
        for (int j = 0; j < count; ++j) { muweight_diff[j] *= this->masks_[j]; }
        if (this->pruned_ratio < this->prune_ratio) {
            if (mthd == "Prune" && APP::criteria == "diff") {
                /// UpdateMasks(); 
            } else if (mthd == "TP" && (APP::step_ - 1) % GetPruneInterval() == 0) {
                TaylorPrune(top);
            }
        }
    }
    


/// ------------------------------------------------------------- 
  
  
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
