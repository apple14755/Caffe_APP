#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/layers/conv_layer.hpp"
#include "caffe/adaptive_probabilistic_pruning.hpp"
#define SHOW_INTERVAL 10
#define SHOW_NUM_LAYER 5
#define LAYER_PRINTED 0

using namespace std;
namespace caffe {

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
          
    /// ADDED BY WANGHUAN -----------------------------------
    Dtype* muweight = this->blobs_[0]->mutable_cpu_data();
    const int count = this->blobs_[0]->count();
    const int num_row = this->blobs_[0]->shape()[0];
    const int num_col = count / num_row;
    const string layer_name = this->layer_param_.name();
    const string mthd = APP::prune_method;
    const int L = APP::layer_index[layer_name];
    this->IF_restore = false;
    
    /// IF_mask
    const bool IF_prune       = mthd != "None";
    const bool IF_enough_iter = (APP::step_ - 1) >= APP::prune_begin_iter;
    const bool IF_pruned      = APP::pruned_ratio[L] > 0;
    this->IF_mask             = IF_prune && (IF_enough_iter || IF_pruned);
    
    
    if (this->phase_ == TRAIN) {
        if (this->IF_mask) {
            if (APP::IF_update_row_col) {
                // UpdateNumPrunedRow/Col
                // Note that, UpdateNumPrunedRow/Col before pruning, 
                // so that when calculating score, the zombie weights will not be counted.
                if ((mthd.substr(0, 3) == "PPc" || mthd == "Reg_Col") && L != APP::layer_cnt-1) {
                    if (APP::step_-1 - APP::iter_prune_finished[L+1] <= 1) {
                        UpdateNumPrunedRow();
                    }
                } else if ((mthd == "PPr" || mthd == "FP") && L != 0 && APP::pruned_rows.size()) {
                    cout << "pruned_rows.size(): " << APP::pruned_rows.size() << endl;
                    UpdateNumPrunedCol();
                } /// Note we don't update column for TP, because their method didn't mention this.
            }
            UpdatePrunedRatio();
            
            // Given pruned_ratio, judge whether prune finished for current layer
            // Get into here ONLY once
            if (APP::iter_prune_finished[L] == INT_MAX) {
                const Dtype pruned_ratio = (mthd == "PPr" || mthd == "FP" || mthd == "TP") ? APP::pruned_ratio_row[L] : APP::pruned_ratio_col[L];
                const bool layer_finish = (pruned_ratio >= APP::prune_ratio[L]); /// layer pruning target achieved
                const bool net_finish = APP::IF_speedup_achieved; /// net pruning target achieved
                if (layer_finish || net_finish) {
                    if (mthd.substr(0, 2) == "PP") { CleanWorkForPP(); } /// last time, do some clean work
                    APP::iter_prune_finished[L] = APP::step_ - 1;
                    char rlayer[10];
                    char rrow[10];
                    char rcol[10];
                    sprintf(rlayer, "%6.4f", APP::pruned_ratio[L]);
                    sprintf(rrow,   "%6.4f", APP::pruned_ratio_row[L]);
                    sprintf(rcol,   "%6.4f", APP::pruned_ratio_col[L]);
                    cout << layer_name << " prune finished!" 
                         << "  step: " << APP::step_
                         << "  speedup: " << APP::speedup
                         << "  pruned_ratio_row: " << rrow
                         << "  pruned_ratio_col: " << rcol 
                         << "  prune_ratio: " << APP::prune_ratio[L] << endl;
                    IF_alpf();
                }
            }
        }
        
        // Print and check, before update probs
        if (L == LAYER_PRINTED && APP::step_ % SHOW_INTERVAL == 0 && APP::inner_iter == 0) {
            Print(L, 'f');
        }

        // Update masks and apply masks
        if (this->IF_mask && APP::iter_prune_finished[L] == INT_MAX) {
            if (mthd == "FP" && (APP::step_ - 1) % APP::prune_interval == 0) {
                FilterPrune(); 
            } else if (mthd.substr(0, 3) == "PPc" && IF_hppf()) {
                if (APP::prune_interval) {
                    ProbPruneCol(APP::prune_interval);
                } else {
                    ProbPruneCol();
                }
            } else if (mthd == "PPr" && IF_hppf()) {
                if (APP::prune_interval) {
                    ProbPruneRow(APP::prune_interval);
                }
            } else if (mthd == "Reg_Col") {
                PruneMinimals(APP::prune_threshold);
            }
        }
        UpdatePrunedRatio();
        
        // Print 
        if (mthd != "None" && L < SHOW_NUM_LAYER && APP::inner_iter == 0) {
            cout << layer_name << "  IF_mask: " << this->IF_mask 
                 << "  pruned_ratio: " << APP::pruned_ratio[L];
            if (mthd == "PPr" || mthd == "FP" || mthd == "TP") {
                cout << "  pruned_ratio_col: " << APP::num_pruned_col[L] * 1.0 / num_col << "(" << APP::num_pruned_col[L] << ")"
                     << "  pruned_ratio_row: " << APP::num_pruned_row[L] * 1.0 / num_row << "(" << APP::num_pruned_row[L] << ")";
            } else {
                cout << "  pruned_ratio_row: " << APP::num_pruned_row[L] * 1.0 / num_row << "(" << APP::num_pruned_row[L] << ")"
                     << "  pruned_ratio_col: " << APP::num_pruned_col[L] * 1.0 / num_col << "(" << APP::num_pruned_col[L] << ")";
            }
            cout << "  prune_ratio: "  << APP::prune_ratio[L] 
                 << "  reg: " << APP::reg_to_distribute[L] 
                 << "/" << ceil(APP::prune_ratio[L] * num_col) * APP::target_reg << endl; 
        }
        
        
        // Weight logging
        if (APP::num_log) {
            const int num_log = APP::log_index[L].size();
            for (int k = 0; k < num_log; ++k) {
                const int index = APP::log_index[L][k];
                Dtype sum = 0;
                for (int i = 0; i < num_row; ++i) {
                    sum += fabs(muweight[i * num_col + index]);
                }
                sum /= num_row;
                APP::log_weight[L][k].push_back(sum);
            }
        }
    } else {
        if (this->IF_mask && APP::iter_prune_finished[L] == INT_MAX && mthd.substr(0, 2) == "PP") {
            const int num_prune_unit = (mthd == "PPr") ? num_row : num_col;
            Dtype rands[num_prune_unit];
            caffe_rng_uniform(num_prune_unit, (Dtype)0, (Dtype)1, rands);
            for (int i = 0; i < count; ++i) {
                const int row_index = i / num_col;
                const int col_index = i % num_col;
                const bool cond1 = (mthd == "PPr") ? rands[row_index] < APP::history_prob[L][row_index]
                                                   : rands[col_index] < APP::history_prob[L][col_index];
                const bool cond2 = (mthd == "PPr") ? !APP::IF_col_pruned[L][col_index][0]
                                                   : !APP::IF_row_pruned[L][row_index];
                APP::masks[L][i] = (cond1 && cond2) ? 1 : 0;
                this->weight_backup[i] = muweight[i]; // backup weights
                muweight[i] *= APP::masks[L][i];
            }
            this->IF_restore = true;
        }
    }
  /// ------------------------------------------------------
  
    const Dtype* weight = this->blobs_[0]->gpu_data();
    for (int i = 0; i < bottom.size(); ++i) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Dtype* top_data = top[i]->mutable_gpu_data();
        for (int n = 0; n < this->num_; ++n) {
            this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
            top_data + n * this->top_dim_);
            if (this->bias_term_) {
                const Dtype* bias = this->blobs_[1]->gpu_data();
                this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
            }
        }
    }
    /// this->bottom_dim_: bottom feature map size, input
    /// this->top_dim_: top feature map size, output
    /// this->num_: batch size
    
    /// Print feature map to check --------
    /// If row 3 and 8 are pruned in previous layer, then channel 3 and 8 will be only biases in this layer's feature map.
    /**
    if (!APP::IN_TEST && L == 0) {
        cout << "bottom.size(): " << bottom.size() << endl;
        for (int i = 0; i < bottom.size(); ++i) {
            const Dtype* top_data = top[i]->cpu_data();
            const int channel = top[i]->shape()[1];
            const int width   = top[i]->shape()[2];
            const int height  = top[i]->shape()[3];
            cout << "channel: " << channel << " " << width << " " <<  height << endl;
            
            vector<Dtype> sum(channel, 0);
            for (int c = 0; c < channel; ++c) {
                for (int w = 0 ; w < width; ++w) {
                    for (int h = 0; h < height; ++h) {
                        sum[c] += fabs(top_data[0 + c * width * height + w * height + h]);
                    }
                }
            }
            for (int c = 0; c < channel; ++c) {
                cout << sum[c] << "  ";
            }
            cout << endl;
        }
    }
    */
    /// -----------------------------------
    
    
    
    /// Restore weights ----------------
    if (this->IF_restore) {
        /// cout << layer_name << ": restore weights! " << endl;
        this->blobs_[0]->mutable_cpu_data();
        /// this->blobs_[0]->gpu_data(); 
        /// Interesting! If the above line is added, something like "control" seems to transfer from cpu to gpu. 
        /// Then modifying cpu weights won't affect their gpu counterparts.
        for (int i = 0; i < count; ++i) {
            muweight[i] = this->weight_backup[i];
        }
        
        /**
        /// ========================
        /// Chech restore
        cout << "weights from cpu:" << endl;
        for (int i = 0; i < 20; ++i) {
            cout << muweight[i] << " ";
        }
        cout << endl;

        Dtype weight_cpu[count];
        const Dtype* weight_gpu = this->blobs_[0]->gpu_data();
        cout << "weights copied from gpu:" << endl;
        hipMemcpy(weight_cpu, weight_gpu, sizeof(Dtype) * count, hipMemcpyDeviceToHost);
        for (int i = 0; i < 20; ++i) {
            cout << weight_cpu[i] << " ";
        }
        cout << endl;
        /// ========================
        */
    }
    /// --------------------------------
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
  
/// ADDED BY WANGHUAN ------------------------------------------
    Dtype* muweight_diff = this->blobs_[0]->mutable_cpu_diff();      
    const int count   = this->blobs_[0]->count();
    const int num_row = this->blobs_[0]->shape()[0];
    const int num_col = count / num_row;
    const int L = APP::layer_index[this->layer_param_.name()];

    /// Print and check
    if (L == LAYER_PRINTED && APP::step_ % SHOW_INTERVAL == 0 && APP::inner_iter == 0) {
        Print(L, 'b');
    }
    
    /// Diff log
    if (APP::num_log) {
        const int num_log = APP::log_index[L].size();
        for (int i = 0; i < num_log; ++i) {
            const int index = APP::log_index[L][i];
            Dtype sum = 0;
            for (int r = 0; r < num_row; ++r) {
                sum += fabs(muweight_diff[r * num_col + index]);
            }
            sum /= num_row;
            APP::log_diff[L][i].push_back(sum);
        }
    }
    
    if (this->IF_mask) {
        if (APP::iter_prune_finished[L] == INT_MAX) {
            if (APP::prune_method == "TP" && (APP::step_ - 1) % APP::prune_interval == 0) {
                TaylorPrune(top);
            }
        }
        for (int j = 0; j < count; ++j) { 
            muweight_diff[j] *= APP::masks[L][j]; 
        }
    }
/// ------------------------------------------------------------- 
  
  
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
